#include <hipsparse.h>
#include <stdio.h>
#include <stdbool.h>

#define N 18

// error check macros
#define CUSPARSE_CHECK(x) {hipsparseStatus_t _c=x; if (_c != HIPSPARSE_STATUS_SUCCESS) {printf("cusparse fail: %d, line: %d\n", (int)_c, __LINE__); exit(-1);}}

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


int main(int argc, char *argv[]) {

    hipsparseHandle_t hndl;
    hipsparseStatus_t stat;
    hipsparseMatDescr_t descrA;
    hipsparseDirection_t dir = HIPSPARSE_DIRECTION_COLUMN;

    int *nnzPerRow;
    int *csrRowPtrA, *csrColIndA;
    float *csrValA, *d_sparse;
    int nnzA;
    int * nnzTotalDevHostPtr = &nnzA;


    hipMalloc((void**)&nnzPerRow, N*sizeof(int));
    hipMalloc((void**)&csrRowPtrA, N*sizeof(int));
    hipMalloc((void**)&d_sparse, N*N*sizeof(float));
    cudaCheckErrors("hipMalloc fail");

    float h_sparse[N][N]; FILE * f;
    f = fopen("H_matrix", "r");
    int i=0; char x;
    while((x=fgetc(f)) != EOF) {
        if ((x == '1') | (x == '0')) {
           h_sparse[i/N][i%N] = (float)(x-'0');
           i++;
           }
    }
    fclose(f);

//    for (i=0; i<N*N; i++) {
//    if (i%N == 0) printf("\n");
//    printf("%f\t", h_sparse[i/N][i%N]);
//}

    hipMemcpy(d_sparse, h_sparse, N*N*sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy fail");

    CUSPARSE_CHECK(hipsparseCreate(&hndl));
    stat = hipsparseCreateMatDescr(&descrA);
    CUSPARSE_CHECK(stat);
    stat = hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    CUSPARSE_CHECK(stat);
    stat = hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    CUSPARSE_CHECK(stat);
    hipsparseSnnz(hndl, dir, N, N,
                descrA, d_sparse, N,
                nnzPerRow, nnzTotalDevHostPtr);
    if (nnzTotalDevHostPtr != NULL) {
        nnzA = * nnzTotalDevHostPtr;
    }

    int h_nnzPerRow[N];
    hipMemcpy(h_nnzPerRow, nnzPerRow, sizeof(int)*N, hipMemcpyDeviceToHost);
    for (int i=0; i<N; i++) {
        printf("%d\n", h_nnzPerRow[i]);
    }

    hipMalloc((void**)&csrValA, sizeof(float)*nnzA);
    hipMalloc((void**)&csrColIndA, sizeof(int)*nnzA);
    cudaCheckErrors("hipMalloc fail");
    hipsparseSdense2csr(hndl, N, N,
                       descrA, d_sparse,
                       N, nnzPerRow,
                       csrValA, csrRowPtrA, csrColIndA);

    float * h_csrValA; int * h_csrRowPtrA, * h_csrColIndA;
    h_csrValA = (float *)malloc(nnzA*sizeof(float));
    h_csrColIndA = (int *)malloc(nnzA*sizeof(int));
    h_csrRowPtrA = (int *)malloc(N*sizeof(int));
    hipMemcpy(h_csrValA, csrValA, nnzA*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_csrColIndA, csrColIndA, nnzA*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_csrRowPtrA, csrRowPtrA, N*sizeof(float), hipMemcpyDeviceToHost);    
    for (int i=0;i<nnzA;i++) {
        printf("\n%f\t%d", h_csrValA[i], h_csrColIndA[i]); }
    for (int i =0; i<N; i++) {
        printf("\n%d", h_csrRowPtrA[i]); }


    float h_X[N][N], *h_Y, *X, *Y;
    h_Y = (float *)malloc(N*N*sizeof(float));
    for (int i=0; i<N; i++) {
        for (int j=0; j<N; j++) {
            h_X[i][j] = (i==j) ? 1 : 0;
        }
    }

//   for (int i=0; i<N; i++) {
//       printf("\n");
//       for (int j=0; j<N; j++) {
//            printf("%f\t", h_X[i][j]);
//        }
//    }

    hipMalloc((void**)&X, N*N*sizeof(float));
    hipMalloc((void**)&Y, N*N*sizeof(float));
    cudaCheckErrors("hipMalloc fail");
    hipMemcpy(X, h_X, N*N*sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy fail");

    hipsparseOperation_t operationA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    cusparseSolveAnalysisInfo_t info;
    stat = cusparseCreateSolveAnalysisInfo(&info);
    CUSPARSE_CHECK(stat);
    stat = cusparseScsrsm_analysis(hndl, operationA, N, nnzA,
                            descrA, csrValA, csrRowPtrA, csrColIndA,
                            info);
    CUSPARSE_CHECK(stat);

    float p = 1;
    const float *alpha = &p;
    stat = cusparseScsrsm_solve(hndl, operationA, N, N,
                                alpha,
                                descrA,
                                csrValA, csrRowPtrA, csrColIndA,
                                info, X, N, Y, N);
    CUSPARSE_CHECK(stat);

    hipMemcpy(h_Y, Y, N*N*sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy fail");

//    for(int i=0; i<N*N; i++) {
//        if (i%N==0) printf("\n");
//        printf("%f\t", h_Y[i]);
//}
}
